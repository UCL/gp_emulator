#include "hip/hip_runtime.h"
/*********************************************//**
 * vector scalar operation, update vec by,
 * vec = scalar - vec
 *********************************************/
#include "gpu_predict.h"
__global__
void kernel_scalarMinusVec(real *vec, const real scalar, const int size)
{
    int ix = blockIdx.x * blockDim.x + threadIdx.x;
    if( ix < size )
        vec[ix] = scalar - vec[ix];
}


void gpu_scalarMinusVec( real *matrix, const real scalar, const int size )
{
    int nthread, nblock;
    if( size > 65536 * 1024 )
    {
        printf("gpu_scalarMinusVec: size = %d [ > 65536 * 1024 ].\n", size);
        exit(EXIT_FAILURE);
    }
    
    if( size < 1024 )
    {
        nthread = size;
        nblock = 1;
    }
    else
    {
        nthread = 1024;
        nblock = ceil( float(size) / float(nthread) );
    }

    kernel_scalarMinusVec<<<nblock,nthread>>>(matrix, scalar, size);
}


