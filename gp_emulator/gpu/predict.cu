#include "hip/hip_runtime.h"
#include "gpu_predict.h"
#include <stdlib.h>
//#define IDX2C(i,j,ld) (((j)*(ld))+(i))
#define IDX2D(i,j,ld) (((j)*(ld))+(i))  //!! keep it in column major for coping with cublas column major fashion.
#define debug 
// x -> i -> col
// y -> j -> row
// leading dimension should always be column
/*********************************************//** 
 * vector matrix elementwise multiplication
 *********************************************/
__global__ 
void gpu_vectorTimesMatrix(const real *A, const real * v, real *res, int A_ld)
{
    int ix, iy;
    ix = blockIdx.x * blockDim.x + threadIdx.x;
    iy = blockIdx.y * blockDim.y + threadIdx.y;
    res[IDX2D(ix, iy, A_ld)] = A[IDX2D(ix, iy, A_ld)] * v[iy];
}


// ix -> M; iy -> N; iz -> D

__global__
void gpu_cdist(const real *input1, const real *input2, real *output, int In1_ld, int In2_ld, int Out_ld, int D)
{
    int ix, iy, iz;
    ix = blockIdx.x * blockDim.x + threadIdx.x;//N
    iy = blockIdx.y * blockDim.y + threadIdx.y;//M
    iz = blockIdx.z * blockDim.z + threadIdx.z;
    output[IDX2D(ix, iy, Out_ld)] += pow(input1[IDX2D(iy, iz, In1_ld)] - input2[IDX2D(ix, iz, In2_ld)],2);
}


__global__
void gpu_init_array(real *vec, const real init_val)
{
    int ix = blockIdx.x * blockDim.x + threadIdx.x;
    vec[ix] = init_val;
}
// further tests needed: ix exceed M*N
__global__
void gpu_matrixExp(real *matrix, real alpha, real beta)
{
    int ix = blockIdx.x * blockDim.x + threadIdx.x;
    matrix[ix] = beta * exp( alpha * matrix[ix]);
}


void computeTranspose(real *matrix, const  int size_in, const  int size_out)
{
    real * temp;
    temp = ( real *)malloc(sizeof(real) * size_in * size_out);

    for ( int i = 0; i < size_in * size_out; ++i)
        temp[i] = matrix[i];

    for (int y = 0; y < size_out; ++y)
    {
        for (int x = 0; x < size_in; ++x)
        {
            matrix[(x * size_out) + y] = temp[(y * size_in) + x];                                                                

        }   
    }
}

__global__
void gpu_elementwiseMult(const real *v1, real *v2)
{
    int ix = blockIdx.x * blockDim.x + threadIdx.x;
    v2[ix] = v2[ix] * v1[ix];
}


__global__
void gpu_scalarMinusVec(real *vec, const real scalar)
{
    int ix = blockIdx.x * blockDim.x + threadIdx.x;
    vec[ix] = scalar - vec[ix];
}


real* gpu_rowSum(const real *A, const int A_nrows,const int A_ncols)
{
    hipblasStatus_t stat;
    hipblasHandle_t handle;
    stat=hipblasCreate(&handle);
    
    real alpha = 1.f;
    real beta = 0.f;
    real *vec_one;
    real *d_var;
    
    hipMalloc((void **)&vec_one, sizeof(real) * A_ncols );
    hipMalloc((void **)&d_var, sizeof(real) * A_ncols);
    
    gpu_init_array<<< ceil(float(A_ncols)/512), 512 >>>(vec_one, 1);
    gpu_init_array<<< ceil(float(A_ncols)/512),512 >>>(d_var, 0);

    cublasCheckErrors(hipblasDgemv(handle, HIPBLAS_OP_T, A_nrows, A_ncols, &alpha, A, A_nrows, vec_one, 1, &beta, d_var, 1));
    
    hipFree(vec_one);
    hipblasDestroy(handle);
    return d_var;
}

extern "C"{
void predict(real *c_theta_exp, real *c_inputs,real *c_invQt,real *c_invQ, real *c_testing,  int N, int M, int  D, int theta_size)
{
    printf("start Gaussian process prediction: (N=%d,nn=%d,D=%d,theta_size=%d)\n",N,M,D,theta_size);   
    int i;
    hipblasStatus_t stat;
    hipblasHandle_t handle;
    
    stat=hipblasCreate(&handle);

    real *c_theta_exp_sqrt;
    c_theta_exp_sqrt = (real *)malloc( sizeof(real) * theta_size );
    for( i=0; i < theta_size; i++ )
    {
        c_theta_exp_sqrt[i] = sqrt( c_theta_exp[i] );
    }


    //define device vector and matrices
    real *d_inputs, *d_theta_exp, *d_theta_exp_sqrt, *d_invQt, *d_invQ, *d_testing;
    real *d_a;//, *d_deriv;


    //allocate and copy vector on device 
    hipMalloc( (void **)&d_theta_exp, sizeof(real) * theta_size );
    hipMalloc( (void **)&d_theta_exp_sqrt, sizeof(real) * theta_size );
    hipMalloc( (void **)&d_invQt, sizeof(real) * M);
    cublasCheckErrors(hipblasSetVector( theta_size, sizeof(real), c_theta_exp_sqrt, 1, d_theta_exp_sqrt, 1 ));
    cublasCheckErrors(hipblasSetVector( theta_size, sizeof(real), c_theta_exp, 1, d_theta_exp, 1 ));
    cublasCheckErrors(hipblasSetVector( M, sizeof(real), c_invQt, 1, d_invQt, 1));

    //allocate and copy matrix on device
    hipMalloc( (void **)&d_inputs, sizeof(real) * M * D );
    hipMalloc( (void **)&d_invQ, sizeof(real) * M * M );
    hipMalloc( (void **)&d_testing, sizeof(real) * N * D );
    cublasCheckErrors(hipblasSetMatrix( M, D, sizeof(real), c_inputs, M, d_inputs, M ));
    cublasCheckErrors(hipblasSetMatrix( M, M, sizeof(real), c_invQ, M, d_invQ, M ));
    cublasCheckErrors(hipblasSetMatrix( N, D, sizeof(real), c_testing, N, d_testing, N));
    
    //allocate memory to results matrices
    hipMalloc((void **)&d_a, sizeof(real) * M * N);

        
    /*********************
     *cdist
     *********************/
    real *d_res_temp1, *d_res_temp2;
    hipMalloc((void **)&d_res_temp1, sizeof(real) * M * D);
    hipMalloc((void **)&d_res_temp2, sizeof(real) * N * D);
    


    dim3 nthread(1,D);
    dim3 nblock(M,1);
    gpu_vectorTimesMatrix<<<nblock, nthread>>>(d_inputs, d_theta_exp_sqrt, d_res_temp1, M);
    nthread.x=1; nthread.y=D;
    nblock.x=N; nblock.y=1;
    gpu_vectorTimesMatrix<<<nblock, nthread>>>(d_testing, d_theta_exp_sqrt  , d_res_temp2, N);
    gpu_init_array<<<ceil(float(N)*float(M)/512),512>>>(d_a, 0);

    nthread.x=1;   nthread.y=5;    nthread.z=1;
    nblock.x=N;    nblock.y=M/5;     nblock.z=D;
    gpu_cdist<<<nblock,nthread>>>(d_res_temp1, d_res_temp2, d_a, M, N, N, D);

    gpu_matrixExp<<<ceil(float(M)*float(N)/512),512>>>(d_a, -0.5, c_theta_exp[D]);
   
    
    real *d_mu;
    hipMalloc((void **)&d_mu, sizeof(real) * N);
    real alpha = 1.f;
    real beta = 0.f;

   // if( sizeof(real) == sizeof(float) )
   //     cublasCheckErrors(hipblasSgemv(handle, HIPBLAS_OP_N, N, M, &alpha, d_a, N, d_invQt, 1, &beta, d_mu, 1));
    if( sizeof(real) == sizeof(double) )
        cublasCheckErrors(hipblasDgemv(handle, HIPBLAS_OP_N, N, M, &alpha, d_a, N, d_invQt, 1, &beta, d_mu, 1));
    
    real *temp_dot, *d_a_T;
    hipMalloc((void **)&temp_dot, sizeof(real) * M * N);
    hipMalloc((void **)&d_a_T, sizeof(real) * M * N);

    cublasCheckErrors(hipblasDgemm(handle, HIPBLAS_OP_N, HIPBLAS_OP_T, M, N, M, &alpha, d_invQ, M, d_a, N, &beta, temp_dot, M));
    cublasCheckErrors(hipblasDgeam(handle, HIPBLAS_OP_T, HIPBLAS_OP_N, M, N, &alpha, d_a, N, &beta, d_a, M, d_a_T, M));
    gpu_elementwiseMult<<< N, M >>>(d_a_T, temp_dot);
    real *d_var;
    d_var = gpu_rowSum(temp_dot, M, N);
    gpu_scalarMinusVec<<< N, M >>>(d_var, c_theta_exp[D]);







#define debug            
#ifdef debug
    real *temp_;
    temp_ = (real *)malloc( sizeof(real) * N * M);
    //hipblasGetMatrix(M, N, sizeof(real), temp_dot, M, temp_,M);
    hipMemcpy(temp_, d_var, sizeof(real)* N, hipMemcpyDeviceToHost);
//    computeTranspose(temp_, M, N);
    printf("b=%f\n", c_theta_exp[D]);
    for( i = 0; i < 10 ; ++i )
       printf("%.4f|", temp_[i]);
#endif
    
    hipFree(d_var); 
    hipFree(d_theta_exp);
    hipFree(d_testing);
}
}
