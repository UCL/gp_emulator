#include "hip/hip_runtime.h"
/*********************************************//** 
 * CUDA implementation of derived from GaussianProcess
 * in python code. 
 * 
 * Sinan Shi (UCL) 
 *********************************************/
#include "gpu_predict.h"
#include <stdlib.h>
#define debug 
/*********************************************//**
 * predict function:
 * This is the corresponding CUDA function of
 * GaussianPredict:predict in python code.
 *********************************************/
extern "C"{
void predict(const real *c_theta_exp, const real *c_inputs,const real *c_invQt,const real *c_invQ, const real *c_testing,  
        real *c_mu, real *c_var, real *c_deriv,const int N,const int M, const int  D, const int theta_size)
{
    int i,j;
    hipblasStatus_t stat;
    hipblasHandle_t handle;
    
    stat=hipblasCreate(&handle);

    real *c_theta_exp_sqrt;
    c_theta_exp_sqrt = (real *)malloc( sizeof(real) * theta_size );
    for( i=0; i < theta_size; i++ )
    {
        c_theta_exp_sqrt[i] = sqrt( c_theta_exp[i] );
    }


    //define device vector and matrices
    real *d_inputs, *d_theta_exp, *d_theta_exp_sqrt, *d_invQt, *d_invQ, *d_testing;

    //allocate and copy vector on device 
    hipMalloc( (void **)&d_theta_exp, sizeof(real) * theta_size );
    hipMalloc( (void **)&d_theta_exp_sqrt, sizeof(real) * theta_size );
    hipMalloc( (void **)&d_invQt, sizeof(real) * M);
    cublasCheckErrors(hipblasSetVector( theta_size, sizeof(real), c_theta_exp_sqrt, 1, d_theta_exp_sqrt, 1 ));
    cublasCheckErrors(hipblasSetVector( theta_size, sizeof(real), c_theta_exp, 1, d_theta_exp, 1 ));
    cublasCheckErrors(hipblasSetVector( M, sizeof(real), c_invQt, 1, d_invQt, 1));

    //allocate and copy matrix on device
    hipMalloc( (void **)&d_inputs, sizeof(real) * M * D );
    hipMalloc( (void **)&d_invQ, sizeof(real) * M * M );
    hipMalloc( (void **)&d_testing, sizeof(real) * N * D );
    cublasCheckErrors(hipblasSetMatrix( M, D, sizeof(real), c_inputs, M, d_inputs, M ));
    cublasCheckErrors(hipblasSetMatrix( M, M, sizeof(real), c_invQ, M, d_invQ, M ));
    cublasCheckErrors(hipblasSetMatrix( N, D, sizeof(real), c_testing, N, d_testing, N));
    

        
    /*********************************
     * Euclidian distance calculation
     * a = cidist(sqrt(expX_{,D}) * inputs * sqrt(expX_{,D} * testing)))
     * Notice: a is equivalent to a^T in python due to column major fashion
     ********************************/ 
    dim3 nthread, nblock;
    real *d_res_temp1, *d_res_temp2, *d_a;
    hipMalloc((void **)&d_res_temp1, sizeof(real) * M * D);
    hipMalloc((void **)&d_res_temp2, sizeof(real) * N * D);
    hipMalloc((void **)&d_a, sizeof(real) * M * N);

    gpu_vectorTimesMatrix(d_inputs, d_theta_exp_sqrt, d_res_temp1, M, D);
    gpu_vectorTimesMatrix(d_testing, d_theta_exp_sqrt, d_res_temp2, N, D);
    gpu_init_array( d_a, 0.0, N * M );
    gpu_cdist(d_res_temp1, d_res_temp2, d_a, M, D, N, D);
    gpu_matrixExp(d_a, -0.5, c_theta_exp[D], M * N);

    hipFree(d_res_temp1);
    hipFree(d_res_temp2);
int kk;
#undef debug
#ifdef debug
real *h_a;
h_a=(real *)malloc(sizeof(real) * M * N);
hipMemcpy(h_a, d_a, sizeof(real) * M * N, hipMemcpyDeviceToHost);

for(kk =0; kk<10; kk++)
  printf("%f|", h_a[kk]);
  printf("\n");
#endif

    
      

    /*********************************
     * compute mu:
     * mu = a * invQt (dot product)
     ********************************/
    real *d_mu;
    hipMalloc((void **)&d_mu, sizeof(real) * N);
    real alpha = 1.f;
    real beta = 0.f;
    cublasCheckErrors(CUBLAS_GEMV(handle, HIPBLAS_OP_N, N, M, &alpha, d_a, N, d_invQt, 1, &beta, d_mu, 1));
    hipMemcpy(c_mu, d_mu, sizeof(real) * N, hipMemcpyDeviceToHost);
       
    
   /*********************************
    * compute var:
    * var = b - rowsum(a * dot(invQ, a_T))
    ********************************/
    real *d_temp_dot, *d_a_T;
    real *d_var;

    hipMalloc((void **)&d_temp_dot, sizeof(real) * M * N);
    hipMalloc((void **)&d_a_T, sizeof(real) * M * N);

    cublasCheckErrors(CUBLAS_GEMM(handle, HIPBLAS_OP_N, HIPBLAS_OP_T, M, N, M, &alpha, d_invQ, M, d_a, N, &beta, d_temp_dot, M));
    cublasCheckErrors(CUBLAS_GEAM(handle, HIPBLAS_OP_T, HIPBLAS_OP_N, M, N, &alpha, d_a, N, &beta, d_a, M, d_a_T, M));
    
    hipFree(d_a);
    hipFree(d_invQ);
    
    gpu_elementwiseMult(d_a_T, d_temp_dot, M * N);
    d_var = gpu_rowSum(d_temp_dot, M, N);
    gpu_scalarMinusVec(d_var, c_theta_exp[D], N );
    hipMemcpy(c_var, d_var, sizeof(real) * N, hipMemcpyDeviceToHost);

    hipFree(d_var);
    hipFree(d_temp_dot);


   /*********************************
    * compute deriv:
    * 
    ********************************/
    real *d_deriv;
    hipMalloc((void **)&d_deriv, sizeof(real) * N );
    
    nthread.x=1000; nthread.y=1; nthread.z=1;
    nblock.x=ceil(float(N) * float(M) / float(CUDA_BLOCK)/1000); nblock.y=1; nblock.z=1;


    dim3 nblocks_getaa(M, N/1000);
    dim3 nthreads_getaa(1,1000);
    real *d_aa;
    hipMalloc((void **)&d_aa, sizeof(real) * M * N);
    real *ptr_inputs, *ptr_testing, *ptr_deriv;
    ptr_inputs = d_inputs;
    ptr_testing = d_testing;
    ptr_deriv = c_deriv;

     for( i = 0; i < D; ++i){
        gpu_crossMinus(ptr_inputs, ptr_testing, d_aa, M, N );
        ptr_inputs = ptr_inputs + M;
        ptr_testing = ptr_testing + N;
        alpha = c_theta_exp[i];
        gpu_elementwiseMult(d_a_T, d_aa, M * N);
        cublasCheckErrors(CUBLAS_GEMV(handle, HIPBLAS_OP_T, M, N, &alpha, d_aa, M, d_invQt, 1, &beta, d_deriv,1));

        hipMemcpy(ptr_deriv, d_deriv, sizeof(real) * N, hipMemcpyDeviceToHost);
     }

     hipFree(d_mu);
     hipFree(d_invQt);
     hipFree(d_a_T);
     hipFree(d_aa);
     hipFree(d_inputs);
     hipFree(d_deriv);
     hipFree(d_theta_exp);
     hipFree(d_testing);
   
}
}
