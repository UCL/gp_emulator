#include "gpu_predict.h"


extern "C"{
void predict(real *c_theta_exp, real *c_inputs,real *c_invQt,real *c_invQ, real *c_testing, int N, int NN, int  D, int theta_size)
{
    printf("start Gaussian process prediction: (N=%d,nn=%d,D=%d,theta_size=%d)\n",N,NN,D,theta_size);   

    hipblasStatus_t stat;
    hipblasHandle_t handle;
    stat=hipblasCreate(&handle);

    //define device vector and matrices
    real *d_inputs, *d_theta_exp, *d_invQt, *d_invQ, *d_testing;
    real *d_aa, *d_deriv;


    //allocate and copy vector on device 
    hipMalloc( (void **)&d_theta_exp, sizeof(real) * theta_size );
    hipMalloc( (void **)&d_invQt, sizeof(real) * NN);
    cublasCheckErrors(hipblasSetVector( theta_size, sizeof(real), c_theta_exp, 1, d_theta_exp, 1 ));
    cublasCheckErrors(hipblasSetVector( NN, sizeof(real), c_invQt, 1, d_invQt, 1));


    //allocate and copy matrix on device
    hipMalloc( (void **)&d_inputs, sizeof(real) * NN * D );
    hipMalloc( (void **)&d_invQ, sizeof(real) * NN * NN );
    hipMalloc( (void **)&d_testing, sizeof(real) * N * D );
    cublasCheckErrors(hipblasSetMatrix( NN, D, sizeof(real), c_inputs, NN, d_inputs, NN ));
    cublasCheckErrors(hipblasSetMatrix( NN, NN, sizeof(real), c_invQ, NN, d_invQ, NN ));
    cublasCheckErrors(hipblasSetMatrix( N, D, sizeof(real), c_testing, N, d_testing, N));
    



    
    dim3 threads(10,20); 
    gpu_cdist<<<1,threads>>>(d_testing);    

    cublasCheckErrors(hipblasGetMatrix (N, D, sizeof(real), d_testing, N, c_testing, N));
   
    hipFree(d_theta_exp);
    hipFree(d_testing);
    hipFree(d_aa);
    //free(c_theta);
    //free((char*) c_inputs);
    //return Py_BuildValue ("i",1);
}
}
