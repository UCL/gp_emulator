#include "hip/hip_runtime.h"

/*********************************************//** 
 * CUDA implementation of derived from GaussianProcess
 * in python code. 
 * 
 * Sinan Shi 
 *********************************************/
#include "gpu_predict.h"
#include <stdlib.h>

void gpuPredict::init_gpu(void)
{
    hipblasCreate(&handle);
    hipMalloc( (void **)&d_theta_exp, sizeof(real) * theta_size );
    hipMalloc( (void **)&d_theta_exp_sqrt, sizeof(real) * theta_size );
    hipMalloc( (void **)&d_invQt, sizeof(real) * Ntrain);
    cublasCheckErrors(hipblasSetVector( 
                theta_size, sizeof(real), c_theta_exp_sqrt, 1, d_theta_exp_sqrt, 1 ));
    cublasCheckErrors(hipblasSetVector( 
                theta_size, sizeof(real), c_theta_exp, 1, d_theta_exp, 1 ));
    cublasCheckErrors(hipblasSetVector( 
                Ntrain, sizeof(real), c_invQt, 1, d_invQt, 1));

    //allocate and copy matrix on device
    hipMalloc( (void **)&d_train, sizeof(real) * Ntrain * Ninputs );
    hipMalloc( (void **)&d_invQ, sizeof(real) * Ntrain * Ntrain );
    hipMalloc( (void **)&d_predict, sizeof(real) * Npredict * Ninputs );
    cublasCheckErrors(hipblasSetMatrix( 
                Ntrain, Ninputs, sizeof(real), c_train, Ntrain, d_train, Ntrain ));
    cublasCheckErrors(hipblasSetMatrix( 
                Ntrain, Ntrain, sizeof(real), c_invQ, Ntrain, d_invQ, Ntrain ));
    cublasCheckErrors(hipblasSetMatrix( 
                Npredict, Ninputs, sizeof(real), c_predict, Npredict, d_predict, Npredict));
}

/*********************************//*
 * Euclidian distance calculation
 * 1) res_mv1 = theta_exp_sqrt_{,Ninputs} * train
 * 2) res_mv2 = theta_exp_sqrt_{,Ninputs} * predict
 * 3) dist_matrix = cidist(res_mv1, res_mv2)
 * 4) dist_matrix = -0.5 * exp(expX_{Ninputs})
 * Notice: distance is equivalent to distance^T (a^T) in python
 ********************************/ 
void gpuPredict::compute_distance(void)
{
    real *d_res_mv1, *d_res_mv2;
    hipMalloc((void **)&d_res_mv1, sizeof(real) * Ntrain * Ninputs);
    hipMalloc((void **)&d_res_mv2, sizeof(real) * Npredict * Ninputs);
    hipMalloc((void **)&d_dist_matrix, sizeof(real) * Ntrain * Npredict);

    gpu_vectorTimesMatrix(d_train, d_theta_exp_sqrt, d_res_mv1, Ntrain, Ninputs);
    gpu_vectorTimesMatrix(d_predict, d_theta_exp_sqrt, d_res_mv2, Npredict, Ninputs);
    gpu_init_array( d_dist_matrix, 0.0, Npredict * Ntrain );
    gpu_cdist(d_res_mv1, d_res_mv2, d_dist_matrix, Ntrain, Ninputs, Npredict, Ninputs);
    gpu_matrixExp(d_dist_matrix, -0.5, c_theta_exp[Ninputs], Ntrain * Npredict);

    hipFree(d_res_mv1);
    hipFree(d_res_mv2);
}

/*********************************//*
 * compute result:
 * c_result = dist_matrix * invQt (dot product)
 ********************************/
void gpuPredict::compute_result(void)
{
    real *d_result;
    hipMalloc((void **)&d_result, sizeof(real) * Npredict);
    real alpha = 1.f;
    real beta = 0.f;
    cublasCheckErrors(CUBLAS_GEMV(handle, HIPBLAS_OP_N, Npredict, Ntrain, &alpha, d_dist_matrix, 
                Npredict, d_invQt, 1, &beta, d_result, 1));
    hipMemcpy(c_result, d_result, sizeof(real) * Npredict, hipMemcpyDeviceToHost);
    hipFree(d_result);
}


real * gpuPredict::gpu_transpose(real *d_matrix, const int nrow, const int ncol)
{
    real *d_matrix_T;
    real alpha = 1.f;
    real beta = 0.f;
    hipMalloc((void **)&d_matrix_T, sizeof(real) * nrow * ncol );
    cublasCheckErrors(CUBLAS_GEAM(handle, HIPBLAS_OP_T, HIPBLAS_OP_N, 
                nrow, ncol, &alpha, d_matrix, ncol, &beta, 
                d_matrix, nrow, d_matrix_T, nrow));
    return( d_matrix_T );
}

/*********************************
 * compute error:
 * c_error = b - rowsum(a * dot(invQ, d_dist_matrix_T))
 * arguments d_invQ, d_dist_matrix have been freed.
 ********************************/
void gpuPredict::compute_error()
{
    real alpha = 1.f;
    real beta = 0.f;
    real *d_res_dot;
    real *d_error;
    
    hipMalloc((void **)&d_res_dot, sizeof(real) * Ntrain * Npredict);
    cublasCheckErrors(CUBLAS_GEMM(
                handle, HIPBLAS_OP_N, HIPBLAS_OP_T, 
                Ntrain, Npredict, Ntrain, 
                &alpha, d_invQ, Ntrain, 
                d_dist_matrix, Npredict, 
                &beta, d_res_dot, Ntrain));  // dot(invQ, d_dist_matrix_T)
    d_dist_matrix_T = gpuPredict::gpu_transpose(d_dist_matrix, Ntrain, Npredict);
    gpu_elementwiseMult(d_dist_matrix_T, d_res_dot, Ntrain * Npredict);
    d_error = gpu_rowSum(d_res_dot, Ntrain, Npredict);
    gpu_scalarMinusVec(d_error, c_theta_exp[Ninputs], Npredict );
 
    hipMemcpy(c_error, d_error, sizeof(real) * Npredict, hipMemcpyDeviceToHost);

    hipFree(d_dist_matrix);
    hipFree(d_invQ);
    hipFree(d_error);
    hipFree(d_res_dot);
}



/*********************************//*
 * compute deriv:
 ********************************/
void gpuPredict::compute_deriv( void )
{
    int i;
    real alpha;
    real beta = 0.f;
 
    real *d_deriv, *d_aa;
    hipMalloc((void **)&d_deriv, sizeof(real) * Npredict );
    hipMalloc((void **)&d_aa, sizeof(real) * Ntrain * Npredict);
    real *ptr_train, *ptr_predict, *ptr_deriv;
    ptr_train = d_train;
    ptr_predict = d_predict;
    ptr_deriv = c_deriv;

     for( i = 0; i < Ninputs; ++i){
        gpu_crossMinus(ptr_train, ptr_predict, d_aa, Ntrain, Npredict );
        ptr_train = ptr_train + Ntrain;
        ptr_predict = ptr_predict + Npredict;
        alpha = c_theta_exp[i];
        gpu_elementwiseMult(d_dist_matrix_T, d_aa, Ntrain * Npredict);
        cublasCheckErrors(CUBLAS_GEMV(handle, HIPBLAS_OP_T, Ntrain, Npredict, &alpha, d_aa, Ntrain, d_invQt, 1, &beta, d_deriv,1));

        hipMemcpy(ptr_deriv, d_deriv, sizeof(real) * Npredict, hipMemcpyDeviceToHost);
        ptr_deriv = ptr_deriv + Npredict;
     }

     hipFree(d_deriv);
     hipFree(d_aa);
}

void gpuPredict::free_gpu(void)
{
     hipblasDestroy(handle);
     hipFree(d_invQt);
     hipFree(d_dist_matrix_T);
     hipFree(d_train);
     hipFree(d_theta_exp);
     hipFree(d_predict);
     hipFree(d_theta_exp_sqrt);
}

void gpuPredict::predict( void )
{
     gpuPredict::init_gpu();
     gpuPredict::compute_distance();
     gpuPredict::compute_result();
     gpuPredict::compute_error();
     gpuPredict::compute_deriv();
     gpuPredict::free_gpu();
}






