#include "gpu_predict_test.h"

void testCublasgemm(const real *c_mat1, const real *c_mat2, const real *c_res, 
        const int mat1_nrows, const int mat1_ncols, const int mat2_nrows, 
        const int mat2_ncols)
{
    hipblasHandle_t handle;
    hipblasCreate(&handle);
    

    int i;

    real *d_mat1, *d_mat2, *d_res;
    real *c_gpu_res;
    c_gpu_res = (real *)malloc( sizeof(real) * mat2_nrows * mat2_ncols);
    
    hipMalloc( (void **)&d_mat1, sizeof(real) * mat1_nrows * mat1_ncols );
    hipMalloc( (void **)&d_mat2, sizeof(real) * mat2_nrows * mat2_ncols );
    hipMalloc( (void **)&d_res, sizeof(real) * mat1_nrows * mat2_ncols );


    real *c_mat1_T;
    
    
   c_mat1_T = computeTranspose(c_mat1, mat1_nrows, mat1_ncols); 

    hipMemcpy( d_mat1, c_mat1_T, sizeof(real) * mat1_nrows * mat1_ncols, hipMemcpyHostToDevice);
    hipMemcpy( d_mat2, c_mat2, sizeof(real) * mat1_nrows * mat2_ncols, hipMemcpyHostToDevice);


    real alpha = 1.f;
    real beta = 0.f;
    cublasCheckErrors( CUBLAS_GEMM( handle, HIPBLAS_OP_N, HIPBLAS_OP_T, 
                mat1_nrows, mat2_ncols,  mat1_ncols,
                &alpha, 
                d_mat1, mat1_nrows, 
                d_mat2, mat2_ncols, 
                &beta, 
                d_res, mat1_ncols ));

    hipMemcpy( c_gpu_res, d_res, sizeof(real) * mat1_nrows *  mat2_ncols, hipMemcpyDeviceToHost);
    c_gpu_res = computeTranspose(c_gpu_res, mat1_nrows, mat2_ncols); 
    compare_result( c_gpu_res, c_res, mat1_nrows * mat2_ncols, EPSILON_AVG, EPSILON_MAX, "RESULTS");
    
    free(c_mat1_T);
    hipFree(d_mat1);
    hipFree(d_mat2);
    hipFree(d_res);
    free(c_gpu_res);
}
