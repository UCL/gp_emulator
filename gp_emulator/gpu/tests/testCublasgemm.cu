#include "gpu_predict_test.h"

void testCublasgemm(const real *c_matrix1, const real *c_matrix2, const real *c_result, 
        const int matrix1_nrows, const int matrix1_ncols, const int matrix2_nrows, 
        const int matrix2_ncols)
{
    hipblasHandle_t handle;
    hipblasCreate(&handle);
    

    real *d_matrix1, *d_matrix2, *d_result;
    real *c_gpu_result;
    c_gpu_result = (real *)malloc( sizeof(real) * matrix2_nrows * matrix2_ncols);
    
    hipMalloc( (void **)&d_matrix1, sizeof(real) * matrix1_nrows * matrix1_ncols );
    hipMalloc( (void **)&d_matrix2, sizeof(real) * matrix2_nrows * matrix2_ncols );
    hipMalloc( (void **)&d_result, sizeof(real) * matrix1_nrows * matrix2_ncols );


    real *c_matrix1_T;
    
    
   c_matrix1_T = computeTranspose(c_matrix1, matrix1_nrows, matrix1_ncols); 

    hipMemcpy( d_matrix1, c_matrix1_T, sizeof(real) * matrix1_nrows * matrix1_ncols, hipMemcpyHostToDevice);
    hipMemcpy( d_matrix2, c_matrix2, sizeof(real) * matrix1_nrows * matrix2_ncols, hipMemcpyHostToDevice);


    real alpha = 1.f;
    real beta = 0.f;
    CUBLAS_GEMM( handle, HIPBLAS_OP_N, HIPBLAS_OP_T, 
                matrix1_nrows, matrix2_ncols,  matrix1_ncols,
                &alpha, 
                d_matrix1, matrix1_nrows, 
                d_matrix2, matrix2_ncols, 
                &beta, 
                d_result, matrix1_ncols );

    hipMemcpy( c_gpu_result, d_result, sizeof(real) * matrix1_nrows *  matrix2_ncols, hipMemcpyDeviceToHost);
    c_gpu_result = computeTranspose(c_gpu_result, matrix1_nrows, matrix2_ncols); 
    compare_result( c_gpu_result, c_result, matrix1_nrows * matrix2_ncols, EPSILON_AVG, EPSILON_MAX, "RESULTS");
    
    free(c_matrix1_T);
    hipFree(d_matrix1);
    hipFree(d_matrix2);
    hipFree(d_result);
    free(c_gpu_result);
}
