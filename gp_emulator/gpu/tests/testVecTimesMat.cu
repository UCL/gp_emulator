#include "hip/hip_runtime.h"
#include "gpu_predict_test.h"


void testVecTimesMat(const real *c_vec,const  real *c_mat, const real *c_res,const int vec_len, const int mat_nrows, const int mat_ncols,  const dim3 nblock, const dim3 nthreads)
{
    real *d_vec, *d_mat, *d_res;
    real *c_res_gpu;

    real *c_mat_T;


    CU_ASSERT (vec_len == mat_ncols);
    int i;
    hipMalloc((void **)&d_vec, sizeof(real) * vec_len );
    hipMalloc((void **)&d_mat, sizeof(real) * mat_nrows * mat_ncols );
    hipMalloc((void **)&d_res, sizeof(real) * mat_nrows * mat_ncols );
    
    c_mat_T = (real *)malloc( sizeof(real) * mat_nrows * mat_ncols);
    c_res_gpu = (real *)malloc( sizeof(real) * mat_nrows * mat_ncols );

    for( i = 0; i < mat_nrows * mat_ncols; i++ )
        c_mat_T[i] = c_mat[i];
    computeTranspose(c_mat_T, mat_ncols, mat_nrows);


    cublasCheckErrors(hipblasSetVector( vec_len, sizeof(real), c_vec, 1, d_vec, 1 ));
    cublasCheckErrors(hipblasSetMatrix( mat_nrows, mat_ncols, sizeof(real), c_mat_T, mat_nrows, d_mat, mat_nrows));
    
    gpu_vectorTimesMatrix <<< nblock, nthreads >>> ( d_mat, d_vec, d_res, mat_nrows );

    hipMemcpy(c_res_gpu, d_res, sizeof(real) * mat_nrows * mat_ncols, hipMemcpyDeviceToHost);
    computeTranspose(c_res_gpu, mat_nrows, mat_ncols);
    
    int error = 0;
    for( i = 0; i < mat_nrows * mat_ncols; i++)
    {
        if( abs(c_res[i] - c_res_gpu[i]) > epsilon )
            error++;
    }
    
    CU_ASSERT( error == 0);

    free(c_res_gpu);
    free(c_mat_T);

    hipFree(d_vec);
    hipFree(d_mat);
    hipFree(d_res);

}
    
