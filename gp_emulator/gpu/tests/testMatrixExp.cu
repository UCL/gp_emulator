#include "gpu_predict_test.h"


void testMatrixExp(const real *mat, const real *res, const real alpha,const real beta,const int size)
{
    real *d_mat;
    real *gpu_res;

    gpu_res = (real *)malloc( sizeof(real) * size );
    hipMalloc( (void **)&d_mat, sizeof(real) * size );

    hipMemcpy( d_mat, mat, sizeof(real) * size, hipMemcpyHostToDevice );
    gpu_matrixExp( d_mat, alpha, beta, size );
    hipMemcpy( gpu_res, d_mat, sizeof(real) * size, hipMemcpyDeviceToHost);
    compare_result( gpu_res, res, size, EPSILON_AVG, EPSILON_MAX, "RESULT");
    
    hipFree(d_mat);
    free(gpu_res);
}
    
