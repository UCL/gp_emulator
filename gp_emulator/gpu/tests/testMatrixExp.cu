#include "hip/hip_runtime.h"
#include "gpu_predict_test.h"


void testMatrixExp(const real *mat, const real *res, const real alpha,const real beta,const int size)
{
    int i;
    real error;
    real *d_mat;
    real *gpu_res;

    gpu_res = (real *)malloc(sizeof(real) * size);
    hipMalloc((void **)&d_mat, sizeof(real) * size);

    hipMemcpy(d_mat, mat, sizeof(real) * size, hipMemcpyHostToDevice);
    int nblocks , nthreads;
    nthreads = 512;
    nblocks = ceil( float(size) / nthreads);
    
    gpu_matrixExp<<< nblocks, nthreads  >>> (d_mat, alpha, beta);

    hipMemcpy( gpu_res, d_mat, sizeof(real) * size, hipMemcpyDeviceToHost);
    
    for( i = 0; i < size; i++ )
    {
        error = abs( gpu_res[i] - res[i] );
        CU_ASSERT( error < 1e-6 );
    }

    hipFree(d_mat);
    free(gpu_res);
}
    
