#include "hip/hip_runtime.h"
#include "gpu_predict_test.h"


void testMatrixExp(const real *mat, const real *res, const real alpha,const real beta,const int size)
{
    int i;
    int error = 0;
    real *d_mat;
    real *gpu_res;

    gpu_res = (real *)malloc(sizeof(real) * size);
    hipMalloc((void **)&d_mat, sizeof(real) * size);

    hipMemcpy(d_mat, mat, sizeof(real) * size, hipMemcpyHostToDevice);
//    int nblocks , nthreads;
//    nthreads=1000; 
//    nblocks=ceil(float(size) / float(CUDA_BLOCK) / 1000);
    
    gpu_matrixExp(d_mat, alpha, beta, size);

    hipMemcpy( gpu_res, d_mat, sizeof(real) * size, hipMemcpyDeviceToHost);
    
    for( i = 0; i < size; i++ )
    {
        if(abs( gpu_res[i] - res[i] ) / res[i] > epsilon)
            error++;
    }
    
    if( error != 0)
        printf("MatrixExp error [%d/%d]", error,size);
    CU_ASSERT( error == 0 );
    
    hipFree(d_mat);
    free(gpu_res);
}
    
